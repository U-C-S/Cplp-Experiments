#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>


__global__ void addKernel(int* c, const int* a, const int* b)
{
		int i = threadIdx.x;
		c[i] = a[i] + b[i];
}

int main()
{
		const int arraySize = 5;
		const int a[arraySize] = { 1, 2, 3, 4, 5 };
		const int b[arraySize] = { 10, 20, 30, 40, 50 };
		int c[arraySize] = { 0 };
		int int_size = sizeof(int);

		int* d_a, d_b, d_result;

		hipMalloc(&d_a, arraySize * int_size);
		hipMalloc(&d_b, arraySize * int_size);
		hipMalloc(&d_result, arraySize * int_size);


		// Add vectors in parallel.
		addKernel<<<1, arraySize >>>(c, a, b);

		printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
				c[0], c[1], c[2], c[3], c[4]);

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		hipError_t cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceReset failed!");
				return 1;
		}

		return 0;
}